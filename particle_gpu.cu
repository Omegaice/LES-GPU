#include "hip/hip_runtime.h"
#include "particle_gpu.h"
#include "stdio.h"
#include "assert.h"

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

extern "C" int gpudevices(){
    int nDevices;
    hipGetDeviceCount(&nDevices);
    return nDevices;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void GPUFieldInterpolate( const int GridWidth, const int GridDepth, const double dx, const double dy, const int nnz, const double *z, const double *zz, const int offsetX, const int offsetY, const int offsetZ, const double *uext, const double *vext, const double *wext, const double *Text, const double *T2ext, const int pcount, Particle* particles ){
    for( int idx = 0; idx < pcount; idx++ ){
        int ijpts[2][6];
        ijpts[0][2] = floor(particles[idx].xp[0]/dx) + 1;
        ijpts[1][2] = floor(particles[idx].xp[1]/dy) + 1;

        ijpts[0][1] = ijpts[0][2]-1;
        ijpts[0][0] = ijpts[0][1]-1;
        ijpts[0][3] = ijpts[0][2]+1;
        ijpts[0][4] = ijpts[0][3]+1;
        ijpts[0][5] = ijpts[0][4]+1;

        ijpts[1][1] = ijpts[1][2]-1;
        ijpts[1][0] = ijpts[1][1]-1;
        ijpts[1][3] = ijpts[1][2]+1;
        ijpts[1][4] = ijpts[1][3]+1;
        ijpts[1][5] = ijpts[1][4]+1;

        int kuvpts[6];
        for( int iz = 0; iz < nnz+1; iz++ ){
            if (zz[iz] > particles[idx].xp[2]){
                kuvpts[2] = iz-1;
                break;
            }
        }

        kuvpts[3] = kuvpts[2]+1;
        kuvpts[4] = kuvpts[3]+1;
        kuvpts[5] = kuvpts[4]+1;
        kuvpts[1] = kuvpts[2]-1;
        kuvpts[0] = kuvpts[1]-1;

        int kwpts[6];
        for( int iz = 0; iz < nnz+1; iz++ ){
            if (z[iz] > particles[idx].xp[2]) {
                kwpts[2] = iz-1;
                break;
            }
        }

        kwpts[3] = kwpts[2]+1;
        kwpts[4] = kwpts[3]+1;
        kwpts[5] = kwpts[4]+1;
        kwpts[1] = kwpts[2]-1;
        kwpts[0] = kwpts[1]-1;

        double wt[4][6] = {
            { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 },
            { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 },
            { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 },
            { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 },
        };

        double dxvec[2] = { dx, dy };
        for( int iz = 0; iz < 2; iz++ ){
            for( int j = 0; j < 6; j++ ){
                double xjval = dxvec[iz]*(ijpts[iz][j]-1);
                double pj = 1.0;
                for( int k = 0; k < 6; k++ ){
                    double xkval = dxvec[iz]*(ijpts[iz][k]-1);
                    if (j != k) {
                        pj = pj*(particles[idx].xp[iz]-xkval)/(xjval-xkval);
                    }
                }
                wt[iz][j] = pj;
            }
        }

      int first, last;
       if (kuvpts[2] == 1) {
          first = 3;
          last = 4;
          kuvpts[0] = 1;
          kuvpts[1] = 1;
       } else if (kuvpts[2] == 0) {
          first = 4;
          last = 5;
          kuvpts[0] = 1;
          kuvpts[1] = 1;
          kuvpts[2] = 1;
       } else if (kuvpts[2] < 0) {
          first = 0;
          last = 0;
       } else if (kuvpts[2] == 2) {
          first = 2;
          last = 5;
       } else if (kuvpts[2] == nnz) {
          first = 2;
          last = 3;
          kuvpts[3] = nnz;
          kuvpts[4] = nnz;
          kuvpts[5] = nnz;
       } else if (kuvpts[2] > nnz) {
          first = 0;
          last = 0;
       } else if (kuvpts[2] == nnz-1) {
          first = 3;
          last = 4;
          kuvpts[4] = nnz;
          kuvpts[5] = nnz;
       } else if (kuvpts[2] == nnz-2) {
          first = 2;
          last = 5;
       } else {
          first = 1;
          last = 6;
       }

       for( int j = first-1; j < last; j++){
           double xjval = zz[kuvpts[j]];
           double pj = 1.0;
           for( int k = first-1; k < last; k++ ){
              double xkval = zz[kuvpts[k]];
              if (j != k) {
                 pj = pj*(particles[idx].xp[2]-xkval)/(xjval-xkval);
              }
           }
           wt[2][j] = pj;
       }

       if (kwpts[2] == 0) {
          first = 3;
          last = 4;
       } else if (kwpts[2] < 0) {
          first = 0;
          last = 0;
       } else if (kwpts[2] == 1) {
          first = 2;
          last = 5;
       } else if (kwpts[2] == nnz-1) {
          first = 3;
          last = 4;
       } else if (kwpts[2] > nnz) {
          first = 0;
          last = 0;
       } else if (kwpts[2] == nnz-2) {
          first = 2;
          last = 5;
       } else {
          first = 1;
          last = 6;
       }

       for( int j = first-1; j < last; j++){
           double xjval = z[kwpts[j]];
           double pj = 1.0;
           for( int k = first-1; k < last; k++ ){
              double xkval = z[kwpts[k]];
              if (j != k){
                 pj = pj*(particles[idx].xp[2]-xkval)/(xjval-xkval);
              }
           }
           wt[3][j] = pj;
       }

        particles[idx].uf[0] = 0.0;
        particles[idx].uf[1] = 0.0;
        particles[idx].uf[2] = 0.0;

        particles[idx].Tf = 0.0;
        particles[idx].qinf = 0.0;
        for( int k = 0; k < 6; k++ ){
            for( int j = 0; j < 6; j++ ){
                for( int i = 0; i < 6; i++ ){
                    const int ix = ijpts[0][i] + offsetZ - 1;
                    const int iy = ijpts[1][j] + offsetY - 1;
                    const int izuv = kuvpts[k] + offsetX - 1;
                    const int izw = kwpts[k] + offsetX - 1;

                    particles[idx].uf[0] = particles[idx].uf[0]+uext[iy*GridWidth+ix*GridWidth*GridDepth+izuv]*wt[0][i]*wt[1][j]*wt[2][k];
                    particles[idx].uf[1] = particles[idx].uf[1]+vext[iy*GridWidth+ix*GridWidth*GridDepth+izuv]*wt[0][i]*wt[1][j]*wt[2][k];
                    particles[idx].uf[2] = particles[idx].uf[2]+wext[iy*GridWidth+ix*GridWidth*GridDepth+izw]*wt[0][i]*wt[1][j]*wt[3][k];
                    particles[idx].Tf = particles[idx].Tf+Text[iy*GridWidth+ix*GridWidth*GridDepth+izuv]*wt[0][i]*wt[1][j]*wt[2][k];
                    particles[idx].qinf = particles[idx].qinf+T2ext[iy*GridWidth+ix*GridWidth*GridDepth+izuv]*wt[0][i]*wt[1][j]*wt[2][k];
                }
            }
        }
    }
}

__global__ void GPUUpdateParticles( const int it, const int stage, const double dt, const int pcount, Particle* particles ) {
    const double ievap = 1;

	const double Gam = 7.28 * std::pow( 10.0, -2 );
	const double Ion = 2.0;
	const double Os = 1.093;
	const double rhoa = 1.1;
	const double rhow = 1000.0;
	const double nuf  = 1.537e-5;
	const double pi   = 4.0 * std::atan( 1.0 );
	const double pi2  = 2.0 * pi;
	const double Sal = 34.0;
	const double radius_mass = 40.0e-6;
	const double m_s = Sal / 1000.0 * 4.0 / 3.0 * pi * std::pow(radius_mass, 3) * rhow;
    const double Pra = 0.715;
    const double Sc = 0.615;
    const double Mw = 0.018015;
    const double Ru = 8.3144;
    const double Ms = 0.05844;
    const double Cpa = 1006.0;
    const double Cpp = 4179.0;
    const double CpaCpp = Cpa/Cpp;
    const double part_grav = 0.0;

    const double zetas[3] = {0.0, -17.0/60.0, -5.0/12.0};
    const double gama[3]  = {8.0/15.0, 5.0/12.0, 3.0/4.0};
    const double g[3] = {0.0, 0.0, part_grav};

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx > pcount ) return;

    const int istage = stage - 1;
    if( it == 1 ) {
        for( int j = 0; j < 3; j++ ) {
            particles[idx].vp[j] = particles[idx].uf[j];
        }
        particles[idx].Tp = particles[idx].Tf;
    }

    double diff[3];
    for( int j = 0; j < 3; j++ ) {
        diff[j] = particles[idx].vp[j] - particles[idx].uf[j];
    }
    double diffnorm = std::sqrt( diff[0] * diff[0] + diff[1] * diff[1] + diff[2] * diff[2] );
    double Rep = 2.0 * particles[idx].radius * diffnorm / nuf;
    double Volp = pi2 * 2.0 / 3.0 * ( particles[idx].radius * particles[idx].radius * particles[idx].radius);
    double rhop = ( m_s + Volp * rhow ) / Volp;
    double taup_i = 18.0 * rhoa * nuf / rhop / ( (2.0 * particles[idx].radius) * (2.0 * particles[idx].radius) );

    double corrfac = 1.0 + 0.15 * pow( Rep, 0.687 );
    double Nup = 2.0 + 0.6 * pow( Rep, 0.5 ) * pow( Pra, 1.0 / 3.0 );
    double Shp = 2.0 + 0.6 * pow( Rep, 0.5 ) * pow( Sc, 1.0 / 3.0 );

    double TfC = particles[idx].Tf - 273.15;
    double einf = 610.94 * exp( 17.6257 * TfC / ( TfC + 243.04 ) );
    double Lv = ( 25.0 - 0.02274 * 26.0 ) * 100000;
    double Eff_C = 2.0 * Mw * Gam / ( Ru * rhow * particles[idx].radius * particles[idx].Tp );
    double Eff_S = Ion * Os * m_s * Mw / Ms / ( Volp * rhop - m_s );
    double estar = einf * exp( Mw * Lv / Ru * ( 1.0 / particles[idx].Tf - 1.0 / particles[idx].Tp ) + Eff_C - Eff_S );
    particles[idx].qstar = Mw / Ru * estar / particles[idx].Tp / rhoa;

    double xtmp[3], vtmp[3];
    for( int j = 0; j < 3; j++ ) {
        xtmp[j] = particles[idx].xp[j] + dt * zetas[istage] * particles[idx].xrhs[j];
        vtmp[j] = particles[idx].vp[j] + dt * zetas[istage] * particles[idx].vrhs[j];
    }

    double Tptmp = particles[idx].Tp + dt * zetas[istage] * particles[idx].Tprhs_s;
    Tptmp = Tptmp + dt * zetas[istage] * particles[idx].Tprhs_L;
    double radiustmp = particles[idx].radius + dt * zetas[istage] * particles[idx].radrhs;

    for( int j = 0; j < 3; j++ ) {
        particles[idx].xrhs[j] = particles[idx].vp[j];
    }

    for( int j = 0; j < 3; j++ ) {
        particles[idx].vrhs[j] = corrfac * taup_i * (particles[idx].uf[j] - particles[idx].vp[j]) - g[j];
    }

    if( ievap == 1 ) {
        particles[idx].radrhs = Shp / 9.0 / Sc * rhop / rhow * particles[idx].radius * taup_i * ( particles[idx].qinf - particles[idx].qstar );
    } else {
        particles[idx].radrhs = 0.0;
    }

    particles[idx].Tprhs_s = -Nup / 3.0 / Pra * CpaCpp * rhop / rhow * taup_i * ( particles[idx].Tp - particles[idx].Tf );
    particles[idx].Tprhs_L = 3.0 * Lv / Cpp / particles[idx].radius * particles[idx].radrhs;

    for( int j = 0; j < 3; j++ ) {
        particles[idx].xp[j] = xtmp[j] + dt * gama[istage] * particles[idx].xrhs[j];
        particles[idx].vp[j] = vtmp[j] + dt * gama[istage] * particles[idx].vrhs[j];
    }
    particles[idx].Tp = Tptmp + dt * gama[istage] * particles[idx].Tprhs_s;
    particles[idx].Tp = particles[idx].Tp + dt * gama[istage] * particles[idx].Tprhs_L;
    particles[idx].radius = radiustmp + dt * gama[istage] * particles[idx].radrhs;
}

__global__ void GPUUpdateNonperiodic( const double grid_width, const double delta_vis, const int pcount, Particle* particles ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx > pcount ) return;

    const double top = grid_width - delta_vis;
    const double bot = 0.0 + delta_vis;

    if( particles[idx].xp[2] > top ){
        particles[idx].xp[2] = top - (particles[idx].xp[2]-top);
        particles[idx].vp[2] = -particles[idx].vp[2];
    }else if( particles[idx].xp[2] < bot ){
        particles[idx].xp[2] = bot + (bot-particles[idx].xp[2]);
        particles[idx].vp[2] = -particles[idx].vp[2];
    }
}

__global__ void GPUUpdatePeriodic( const double grid_width, const double grid_height, const int pcount, Particle* particles ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx > pcount ) return;

    if( particles[idx].xp[0] > grid_width ){
        particles[idx].xp[0] = particles[idx].xp[0] - grid_width;
    }else if( particles[idx].xp[0] < 0.0 ){
        particles[idx].xp[0] = grid_width + particles[idx].xp[0];
    }

    if( particles[idx].xp[1] > grid_height ){
        particles[idx].xp[1] = particles[idx].xp[1] - grid_height;
    }else if( particles[idx].xp[1] < 0.0 ){
        particles[idx].xp[1] = grid_height + particles[idx].xp[1];
    }
}

extern "C" double rand2(int idum, bool reset) {
      const int NTAB = 32;
      static int iv[NTAB], iy = 0, idum2 = 123456789;
      if( reset ) {
          for( int i = 0; i < NTAB; i++ ){
              iv[i] = 0;
          }
          iy = 0;
          idum2 = 123456789;
      }

      int k = 0, IM1 = 2147483563,IM2 = 2147483399,IMM1 = IM1-1,IA1 = 40014,IA2 = 40692,IQ1 = 53668,IQ2 = 52774,IR1 = 12211,IR2 = 3791, NDIV = 1+IMM1/NTAB;
      double AM = 1.0/IM1,EPS = 1.2e-7,RNMX = 1.0-EPS;

      if( idum <= 0 ){
          idum = MAX(-idum,1);
          idum2 = idum;
          for ( int j = NTAB+8; j > 1; j-- ) {
             k = idum/IQ1;
             idum = IA1*(idum-k*IQ1)-k*IR1;
             if (idum < 0) {
                 idum=idum+IM1;
             }
             if (j <= NTAB) {
                 iv[j] = idum;
             }
          }
          iy = iv[1];
      }

      k=idum/IQ1;
      idum=IA1*(idum-k*IQ1)-k*IR1;
      if (idum < 0) {
          idum=idum+IM1;
        }
      k = idum2/IQ2;
      idum2 = IA2*(idum2-k*IQ2)-k*IR2;
      if (idum2 < 0) {
          idum2=idum2+IM2;
      }
      const int j = 1 + iy/NDIV;
      iy = iv[j] - idum2;
      iv[j] = idum;
      if (iy < 1) {
          iy = iy+IMM1;
      }
      return MIN(AM*iy,RNMX);
}

extern "C" GPU* NewGPU(const int particles, const int width, const int height, const int depth) {
    GPU* retVal = (GPU*) malloc( sizeof(GPU) );

    // Particle Data
    retVal->pCount = particles;
    retVal->hParticles = (Particle*) malloc( sizeof(Particle) * particles );
    gpuErrchk( hipMalloc( (void **)&retVal->dParticles, sizeof(Particle) * retVal->pCount ) );

    // Grid Data
    retVal->GridWidth = width;
    retVal->GridHeight = height;
    retVal->GridDepth = depth;

    return retVal;
}

extern "C" void ParticleAdd( GPU *gpu, const int position, const Particle *input ){
    assert(position >= 0 && position < gpu->pCount);
    assert(input->uf[0] < 20 && input->uf[1] < 20 && input->uf[2] < 20 );
    memcpy(&gpu->hParticles[position], input, sizeof(Particle));
}

extern "C" Particle ParticleGet( GPU *gpu, const int position ){
    assert(position >= 0 && position < gpu->pCount);
    return gpu->hParticles[position];
}

extern "C" void ParticleUpload( GPU *gpu ){
    gpuErrchk( hipMemcpy( gpu->dParticles, gpu->hParticles, sizeof(Particle) * gpu->pCount, hipMemcpyHostToDevice ) );
}

extern "C" void ParticleInit( GPU* gpu, const int particles, const Particle* input ){
    gpu->pCount = particles;
    gpuErrchk( hipMalloc( (void **)&gpu->dParticles, sizeof(Particle) * particles ) );
    gpuErrchk( hipMemcpy( gpu->dParticles, input, sizeof(Particle) * particles, hipMemcpyHostToDevice ) );
}

extern "C" void ParticleGenerate(GPU* gpu, const int processors, const int particles, const int seed, const double temperature, const double xmin, const double xmax, const double ymin, const double ymax, const double zl, const double delta_vis, const double radius, const double qinfp){
    gpu->pCount = particles;
    gpuErrchk( hipMalloc( (void **)&gpu->dParticles, sizeof(Particle) * particles) );

    bool reset = true;
    int currentProcessor = 1;
    const int particles_per_processor = particles / processors;

    Particle *hParticles = (Particle*) malloc( sizeof(Particle) * particles );
    for( size_t i = 0; i < particles; i++ ){
        if( i >= currentProcessor * particles_per_processor) {
            reset = true;
            currentProcessor++;
        }

        double random = 0.0;
        if( reset ) {
            random = rand2(seed, true);
            reset = false;
        }else{
            random = rand2(seed, false);
        }
        const double x = random*(xmax-xmin) + xmin;
        const double y = rand2(seed, false)*(ymax-ymin) + ymin;
        const double z = rand2(seed, false)*(zl-2.0*delta_vis) + delta_vis;

        hParticles[i].xp[0] = x; hParticles[i].xp[1] = y; hParticles[i].xp[2] = z;
        hParticles[i].vp[0] = 0.0; hParticles[i].vp[1] = 0.0; hParticles[i].vp[2] = 0.0;
        hParticles[i].Tp = temperature;
        hParticles[i].radius = radius;
        hParticles[i].uf[0] = 0.0; hParticles[i].uf[1] = 0.0; hParticles[i].uf[2] = 0.0;
        hParticles[i].qinf = qinfp;
        hParticles[i].xrhs[0] = 0.0; hParticles[i].xrhs[1] = 0.0; hParticles[i].xrhs[2] = 0.0;
        hParticles[i].vrhs[0] = 0.0; hParticles[i].vrhs[1] = 0.0; hParticles[i].vrhs[2] = 0.0;
        hParticles[i].Tprhs_s = 0.0;
        hParticles[i].Tprhs_L = 0.0;
        hParticles[i].radrhs = 0.0;
    }

    gpuErrchk( hipMemcpy(gpu->dParticles, hParticles, sizeof(Particle) * particles, hipMemcpyHostToDevice) );
    free(hParticles);
}

extern "C" void ParticleInterpolate( GPU *gpu, const double dx, const double dy, const int nnz, double* z, double *zz, const int offsetX, const int offsetY, const int offsetZ, double *uext, double *vext, double *wext, double *text, double *t2ext ) {
    GPUFieldInterpolate( gpu->GridWidth, gpu->GridHeight, dx, dy, nnz, z, zz, 1-offsetX, 1-offsetY, 1-offsetZ, uext, vext, wext, text, t2ext, gpu->pCount, gpu->hParticles);
}

extern "C" void ParticleStep( GPU *gpu, const int it, const int istage, const double dt ) {
    GPUUpdateParticles<<< (gpu->pCount / 32) + 1, 32 >>> (it, istage, dt, gpu->pCount, gpu->dParticles);
    gpuErrchk( hipPeekAtLastError() );
}

extern "C" void ParticleUpdateNonPeriodic( GPU *gpu, const double grid_width, const double delta_viz ) {
    GPUUpdateNonperiodic<<< (gpu->pCount / 32) + 1, 32 >>> (grid_width, delta_viz, gpu->pCount, gpu->dParticles);
    gpuErrchk( hipPeekAtLastError() );
}

extern "C" void ParticleUpdatePeriodic( GPU *gpu, const double grid_width, const double grid_height ) {
    GPUUpdatePeriodic<<< (gpu->pCount / 32) + 1, 32 >>> (grid_width, grid_height, gpu->pCount, gpu->dParticles);
    gpuErrchk( hipPeekAtLastError() );
}

extern "C" void ParticleDownloadHost( GPU *gpu ) {
    gpuErrchk( hipMemcpy(gpu->hParticles, gpu->dParticles, sizeof(Particle) * gpu->pCount, hipMemcpyDeviceToHost) );
}

extern "C" Particle* ParticleDownload( GPU *gpu ) {
    Particle *result = (Particle*) malloc( sizeof(Particle) * gpu->pCount);
    gpuErrchk( hipMemcpy(result, gpu->dParticles, sizeof(Particle) * gpu->pCount, hipMemcpyDeviceToHost) );
    return result;
}

void ParticleWrite( GPU* gpu ){
    static int call = 0;
    static char buffer[80];
    sprintf(buffer, "c-particle-%d.dat", call);

    FILE *write_ptr = fopen(buffer,"wb");
    call += 1;

    fwrite(&gpu->pCount, sizeof(unsigned int), 1, write_ptr);
    for( int i = 0; i < gpu->pCount; i++ ){
        fwrite(&gpu->hParticles[i], sizeof(Particle), 1, write_ptr);
    }

    fclose(write_ptr);
}

GPU* ParticleRead(char * path){
    FILE *data = fopen(path,"rb");
    GPU *retVal = (GPU*) malloc( sizeof(GPU) );

    fread(&retVal->pCount, sizeof(unsigned int), 1, data);
    retVal->hParticles = (Particle*) malloc( sizeof(Particle) * retVal->pCount );
    for( int i = 0; i < retVal->pCount; i++ ){
        fread(&retVal->hParticles[i], sizeof(Particle), 1, data);
    }

    fclose(data);
    return retVal;
}
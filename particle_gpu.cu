#include "hip/hip_runtime.h"
#include "particle_gpu.h"
#include "stdio.h"
#include "assert.h"

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

extern "C" int gpudevices(){
    int nDevices;
    hipGetDeviceCount(&nDevices);
    return nDevices;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void GPUUpdateParticles( const int it, const int stage, const double dt, const int pcount, Particle* particles ) {
    const double ievap = 1;

	const double Gam = 7.28 * std::pow( 10.0, -2 );
	const double Ion = 2.0;
	const double Os = 1.093;
	const double rhoa = 1.1;
	const double rhow = 1000.0;
	const double nuf  = 1.537e-5;
	const double pi   = 4.0 * std::atan( 1.0 );
	const double pi2  = 2.0 * pi;
	const double Sal = 34.0;
	const double radius_mass = 40.0e-6;
	const double m_s = Sal / 1000.0 * 4.0 / 3.0 * pi * std::pow(radius_mass, 3) * rhow;
    const double Pra = 0.715;
    const double Sc = 0.615;
    const double Mw = 0.018015;
    const double Ru = 8.3144;
    const double Ms = 0.05844;
    const double Cpa = 1006.0;
    const double Cpp = 4179.0;
    const double CpaCpp = Cpa/Cpp;
    const double part_grav = 0.0;

    const double zetas[3] = {0.0, -17.0/60.0, -5.0/12.0};
    const double gama[3]  = {8.0/15.0, 5.0/12.0, 3.0/4.0};
    const double g[3] = {0.0, 0.0, part_grav};

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx > pcount ) return;

    const int istage = stage - 1;
    if( it == 1 ) {
        for( int j = 0; j < 3; j++ ) {
            particles[idx].vp[j] = particles[idx].uf[j];
        }
        particles[idx].Tp = particles[idx].Tf;
    }

    double diff[3];
    for( int j = 0; j < 3; j++ ) {
        diff[j] = particles[idx].vp[j] - particles[idx].uf[j];
    }
    double diffnorm = std::sqrt( diff[0] * diff[0] + diff[1] * diff[1] + diff[2] * diff[2] );
    double Rep = 2.0 * particles[idx].radius * diffnorm / nuf;
    double Volp = pi2 * 2.0 / 3.0 * ( particles[idx].radius * particles[idx].radius * particles[idx].radius);
    double rhop = ( m_s + Volp * rhow ) / Volp;
    double taup_i = 18.0 * rhoa * nuf / rhop / ( (2.0 * particles[idx].radius) * (2.0 * particles[idx].radius) );

    double corrfac = 1.0 + 0.15 * pow( Rep, 0.687 );
    double Nup = 2.0 + 0.6 * pow( Rep, 0.5 ) * pow( Pra, 1.0 / 3.0 );
    double Shp = 2.0 + 0.6 * pow( Rep, 0.5 ) * pow( Sc, 1.0 / 3.0 );

    double TfC = particles[idx].Tf - 273.15;
    double einf = 610.94 * exp( 17.6257 * TfC / ( TfC + 243.04 ) );
    double Lv = ( 25.0 - 0.02274 * 26.0 ) * 100000;
    double Eff_C = 2.0 * Mw * Gam / ( Ru * rhow * particles[idx].radius * particles[idx].Tp );
    double Eff_S = Ion * Os * m_s * Mw / Ms / ( Volp * rhop - m_s );
    double estar = einf * exp( Mw * Lv / Ru * ( 1.0 / particles[idx].Tf - 1.0 / particles[idx].Tp ) + Eff_C - Eff_S );
    particles[idx].qstar = Mw / Ru * estar / particles[idx].Tp / rhoa;

    double xtmp[3], vtmp[3];
    for( int j = 0; j < 3; j++ ) {
        xtmp[j] = particles[idx].xp[j] + dt * zetas[istage] * particles[idx].xrhs[j];
        vtmp[j] = particles[idx].vp[j] + dt * zetas[istage] * particles[idx].vrhs[j];
    }

    double Tptmp = particles[idx].Tp + dt * zetas[istage] * particles[idx].Tprhs_s;
    Tptmp = Tptmp + dt * zetas[istage] * particles[idx].Tprhs_L;
    double radiustmp = particles[idx].radius + dt * zetas[istage] * particles[idx].radrhs;

    for( int j = 0; j < 3; j++ ) {
        particles[idx].xrhs[j] = particles[idx].vp[j];
    }

    for( int j = 0; j < 3; j++ ) {
        particles[idx].vrhs[j] = corrfac * taup_i * (particles[idx].uf[j] - particles[idx].vp[j]) - g[j];
    }

    if( ievap == 1 ) {
        particles[idx].radrhs = Shp / 9.0 / Sc * rhop / rhow * particles[idx].radius * taup_i * ( particles[idx].qinf - particles[idx].qstar );
    } else {
        particles[idx].radrhs = 0.0;
    }

    particles[idx].Tprhs_s = -Nup / 3.0 / Pra * CpaCpp * rhop / rhow * taup_i * ( particles[idx].Tp - particles[idx].Tf );
    particles[idx].Tprhs_L = 3.0 * Lv / Cpp / particles[idx].radius * particles[idx].radrhs;

    for( int j = 0; j < 3; j++ ) {
        particles[idx].xp[j] = xtmp[j] + dt * gama[istage] * particles[idx].xrhs[j];
        particles[idx].vp[j] = vtmp[j] + dt * gama[istage] * particles[idx].vrhs[j];
    }
    particles[idx].Tp = Tptmp + dt * gama[istage] * particles[idx].Tprhs_s;
    particles[idx].Tp = particles[idx].Tp + dt * gama[istage] * particles[idx].Tprhs_L;
    particles[idx].radius = radiustmp + dt * gama[istage] * particles[idx].radrhs;
}

__global__ void GPUUpdateNonperiodic( const double grid_width, const double delta_vis, const int pcount, Particle* particles ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx > pcount ) return;

    const double top = grid_width - delta_vis;
    const double bot = 0.0 + delta_vis;

    if( particles[idx].xp[2] > top ){
        particles[idx].xp[2] = top - (particles[idx].xp[2]-top);
        particles[idx].vp[2] = -particles[idx].vp[2];
    }else if( particles[idx].xp[2] < bot ){
        particles[idx].xp[2] = bot + (bot-particles[idx].xp[2]);
        particles[idx].vp[2] = -particles[idx].vp[2];
    }
}

__global__ void GPUUpdatePeriodic( const double grid_width, const double grid_height, const int pcount, Particle* particles ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx > pcount ) return;

    if( particles[idx].xp[0] > grid_width ){
        particles[idx].xp[0] = particles[idx].xp[0] - grid_width;
    }else if( particles[idx].xp[0] < 0.0 ){
        particles[idx].xp[0] = grid_width + particles[idx].xp[0];
    }

    if( particles[idx].xp[1] > grid_height ){
        particles[idx].xp[1] = particles[idx].xp[1] - grid_height;
    }else if( particles[idx].xp[1] < 0.0 ){
        particles[idx].xp[1] = grid_height + particles[idx].xp[1];
    }
}

extern "C" double rand2(int idum, bool reset) {
      const int NTAB = 32;
      static int iv[NTAB], iy = 0, idum2 = 123456789;
      if( reset ) {
          for( int i = 0; i < NTAB; i++ ){
              iv[i] = 0;
          }
          iy = 0;
          idum2 = 123456789;
      }

      int k = 0, IM1 = 2147483563,IM2 = 2147483399,IMM1 = IM1-1,IA1 = 40014,IA2 = 40692,IQ1 = 53668,IQ2 = 52774,IR1 = 12211,IR2 = 3791, NDIV = 1+IMM1/NTAB;
      double AM = 1.0/IM1,EPS = 1.2e-7,RNMX = 1.0-EPS;

      if( idum <= 0 ){
          idum = MAX(-idum,1);
          idum2 = idum;
          for ( int j = NTAB+8; j > 1; j-- ) {
             k = idum/IQ1;
             idum = IA1*(idum-k*IQ1)-k*IR1;
             if (idum < 0) {
                 idum=idum+IM1;
             }
             if (j <= NTAB) {
                 iv[j] = idum;
             }
          }
          iy = iv[1];
      }

      k=idum/IQ1;
      idum=IA1*(idum-k*IQ1)-k*IR1;
      if (idum < 0) {
          idum=idum+IM1;
        }
      k = idum2/IQ2;
      idum2 = IA2*(idum2-k*IQ2)-k*IR2;
      if (idum2 < 0) {
          idum2=idum2+IM2;
      }
      const int j = 1 + iy/NDIV;
      iy = iv[j] - idum2;
      iv[j] = idum;
      if (iy < 1) {
          iy = iy+IMM1;
      }
      return MIN(AM*iy,RNMX);
}

extern "C" GPU* NewGPU(const int particles) {
    GPU* retVal = (GPU*) malloc( sizeof(GPU) );
    retVal->pCount = particles;
    retVal->hParticles = (Particle*) malloc( sizeof(Particle) * particles );
    gpuErrchk( hipMalloc( (void **)&retVal->dParticles, sizeof(Particle) * retVal->pCount ) );
    return retVal;
}

extern "C" void ParticleAdd( GPU *gpu, const int position, const Particle *input ){
    assert(position >= 0 && position < gpu->pCount);
    assert(input->uf[0] < 20 && input->uf[1] < 20 && input->uf[2] < 20 );
    memcpy(&gpu->hParticles[position], input, sizeof(Particle));
}

extern "C" Particle ParticleGet( GPU *gpu, const int position ){
    assert(position >= 0 && position < gpu->pCount);
    return gpu->hParticles[position];
}

extern "C" void ParticleUpload( GPU *gpu ){
    gpuErrchk( hipMemcpy( gpu->dParticles, gpu->hParticles, sizeof(Particle) * gpu->pCount, hipMemcpyHostToDevice ) );
}

extern "C" void ParticleInit( GPU* gpu, const int particles, const Particle* input ){
    gpu->pCount = particles;
    gpuErrchk( hipMalloc( (void **)&gpu->dParticles, sizeof(Particle) * particles ) );
    gpuErrchk( hipMemcpy( gpu->dParticles, input, sizeof(Particle) * particles, hipMemcpyHostToDevice ) );
}

extern "C" void ParticleGenerate(GPU* gpu, const int processors, const int particles, const int seed, const double temperature, const double xmin, const double xmax, const double ymin, const double ymax, const double zl, const double delta_vis, const double radius, const double qinfp){
    gpu->pCount = particles;
    gpuErrchk( hipMalloc( (void **)&gpu->dParticles, sizeof(Particle) * particles) );

    bool reset = true;
    int currentProcessor = 1;
    const int particles_per_processor = particles / processors;

    Particle *hParticles = (Particle*) malloc( sizeof(Particle) * particles );
    for( size_t i = 0; i < particles; i++ ){
        if( i >= currentProcessor * particles_per_processor) {
            reset = true;
            currentProcessor++;
        }

        double random = 0.0;
        if( reset ) {
            random = rand2(seed, true);
            reset = false;
        }else{
            random = rand2(seed, false);
        }
        const double x = random*(xmax-xmin) + xmin;
        const double y = rand2(seed, false)*(ymax-ymin) + ymin;
        const double z = rand2(seed, false)*(zl-2.0*delta_vis) + delta_vis;

        hParticles[i].xp[0] = x; hParticles[i].xp[1] = y; hParticles[i].xp[2] = z;
        hParticles[i].vp[0] = 0.0; hParticles[i].vp[1] = 0.0; hParticles[i].vp[2] = 0.0;
        hParticles[i].Tp = temperature;
        hParticles[i].radius = radius;
        hParticles[i].uf[0] = 0.0; hParticles[i].uf[1] = 0.0; hParticles[i].uf[2] = 0.0;
        hParticles[i].qinf = qinfp;
        hParticles[i].xrhs[0] = 0.0; hParticles[i].xrhs[1] = 0.0; hParticles[i].xrhs[2] = 0.0;
        hParticles[i].vrhs[0] = 0.0; hParticles[i].vrhs[1] = 0.0; hParticles[i].vrhs[2] = 0.0;
        hParticles[i].Tprhs_s = 0.0;
        hParticles[i].Tprhs_L = 0.0;
        hParticles[i].radrhs = 0.0;
    }

    gpuErrchk( hipMemcpy(gpu->dParticles, hParticles, sizeof(Particle) * particles, hipMemcpyHostToDevice) );
    free(hParticles);
}

extern "C" void ParticleStep( GPU *gpu, const int it, const int istage, const double dt ) {
    GPUUpdateParticles<<< (gpu->pCount / 32) + 1, 32 >>> (it, istage, dt, gpu->pCount, gpu->dParticles);
    gpuErrchk( hipPeekAtLastError() );
}

extern "C" void ParticleUpdateNonPeriodic( GPU *gpu, const double grid_width, const double delta_viz ) {
    GPUUpdateNonperiodic<<< (gpu->pCount / 32) + 1, 32 >>> (grid_width, delta_viz, gpu->pCount, gpu->dParticles);
    gpuErrchk( hipPeekAtLastError() );
}

extern "C" void ParticleUpdatePeriodic( GPU *gpu, const double grid_width, const double grid_height ) {
    GPUUpdatePeriodic<<< (gpu->pCount / 32) + 1, 32 >>> (grid_width, grid_height, gpu->pCount, gpu->dParticles);
    gpuErrchk( hipPeekAtLastError() );
}

extern "C" void ParticleDownloadHost( GPU *gpu ) {
    gpuErrchk( hipMemcpy(gpu->hParticles, gpu->dParticles, sizeof(Particle) * gpu->pCount, hipMemcpyDeviceToHost) );
}

extern "C" Particle* ParticleDownload( GPU *gpu ) {
    Particle *result = (Particle*) malloc( sizeof(Particle) * gpu->pCount);
    gpuErrchk( hipMemcpy(result, gpu->dParticles, sizeof(Particle) * gpu->pCount, hipMemcpyDeviceToHost) );
    return result;
}

void ParticleWrite( GPU* gpu ){
    static int call = 0;
    static char buffer[80];
    sprintf(buffer, "c-particle-%d.dat", call);

    FILE *write_ptr = fopen(buffer,"wb");
    call += 1;

    fwrite(&gpu->pCount, sizeof(unsigned int), 1, write_ptr);
    for( int i = 0; i < gpu->pCount; i++ ){
        fwrite(&gpu->hParticles[i], sizeof(Particle), 1, write_ptr);
    }

    fclose(write_ptr);
}

GPU* ParticleRead(char * path){
    FILE *data = fopen(path,"rb");
    GPU *retVal = (GPU*) malloc( sizeof(GPU) );

    fread(&retVal->pCount, sizeof(unsigned int), 1, data);
    retVal->hParticles = (Particle*) malloc( sizeof(Particle) * retVal->pCount );
    for( int i = 0; i < retVal->pCount; i++ ){
        fread(&retVal->hParticles[i], sizeof(Particle), 1, data);
    }

    fclose(data);
    return retVal;
}